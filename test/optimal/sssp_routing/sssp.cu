#include "hip/hip_runtime.h"
#include "header.h"
#include "util.h"
#include "mapper.cuh"
#include "reducer.cuh"
#include "wtime.h"
#include "barrier.cuh"
#include "gpu_graph.cuh"
#include "meta_data.cuh"
#include "mapper_enactor.cuh"
#include "reducer_enactor.cuh"
#include "cpu_sssp_route.hpp"

/*user defined vertex behavior function*/
__inline__ __host__ __device__ feature_t user_mapper_push(
    vertex_t 	src,
    vertex_t	dest,
    feature_t	level,
    index_t*	beg_pos,
    weight_t	edge_weight,
    feature_t* vert_status,
    feature_t* vert_status_prev
){
	feature_t dist = (vert_status[src]>>32) + edge_weight;
	return (dist<<32) + src;
	//return dist;
} 

/*user defined vertex behavior function*/
__inline__ __host__ __device__ bool vertex_selector_push(
    vertex_t 		vert_id, 
    feature_t 		level,
    vertex_t 		*adj_list, 
    index_t 		*beg_pos, 
    feature_t* 	vert_status,
    feature_t* 	vert_status_prev
){
	return (vert_status[vert_id] != vert_status_prev[vert_id]);
}

/*user defined vertex behavior function*/
__inline__ __host__ __device__ feature_t user_mapper_pull(
    vertex_t 	src,
    vertex_t	dest,
    feature_t	level,
    index_t*	beg_pos,
    weight_t	edge_weight,
    feature_t* vert_status,
    feature_t* vert_status_prev
){
	//return vert_status[src] + edge_weight;
	feature_t dist = (vert_status[src]>>32) + edge_weight;
	return (dist<<32) + src;
	//return dist;
}

/*user defined vertex behavior function*/
__inline__ __host__ __device__ bool vertex_selector_pull(
    vertex_t 	vert_id, 
    feature_t 	level,
    vertex_t* 	adj_list, 
    index_t* 	beg_pos, 
    feature_t* vert_status,
    feature_t* vert_status_prev
){
	return true;
}

__device__ cb_reducer vert_selector_push_d = vertex_selector_push;
__device__ cb_reducer vert_selector_pull_d = vertex_selector_pull;
__device__ cb_mapper vert_behave_push_d = user_mapper_push;
__device__ cb_mapper vert_behave_pull_d = user_mapper_pull;


/*init sssp*/
__global__ void
init(vertex_t src_v, vertex_t vert_count, meta_data mdata)
{
	index_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	while(tid < vert_count) {
		if(tid != src_v) {
			mdata.vert_status[tid] = INFTY;
			mdata.vert_status_prev[tid] = INFTY;
		} else {
			mdata.vert_status[tid] = ((1<<31) - 1);
			//mdata.vert_status[tid] = 1;
			mdata.vert_status_prev[tid] = INFTY;
			
			mdata.worklist_mid[0] = src_v;
			mdata.worklist_sz_sml[0] = 0;
			mdata.worklist_sz_mid[0] = 1;
			mdata.worklist_sz_lrg[0] = 0;
			//mdata.bitmap[src_v>>3] |= (1<<(src_v & 7));
		}
		tid += blockDim.x * gridDim.x;
	}
}

void unpack_cpu_dist(
	feature_t* packed_cpu,
	feature_t* unpacked_dist,
	vertex_t* unpacked_route,
	vertex_t count
){
	for(vertex_t i = 0; i < count; i++) {
		unpacked_dist[i] = (packed_cpu[i] >> 32);
		unpacked_route[i] = (packed_cpu[i] & ((1<<32) - 1));
	}
}

int main(int args, char **argv)
{
    // Based on the high-diameter SSSP
	std::cout<<"Input: /path/to/exe /path/to/beg_pos /path/to/adj_list /path/weight_list src blk_size switch_iter\n";
	if(args<5){
        std::cout<<"Wrong input\n";exit(-1);
    }
    for(int i = 0; i < args; i++) {
		std::cout<<argv[i]<<" ";
	}
    std::cout<<"\n";
		
	double tm_map,tm_red,tm_scan;
	char *file_beg_pos = argv[1];
	char *file_adj_list = argv[2];
	char *file_weight_list = argv[3];
	vertex_t src_v = (vertex_t)atol(argv[4]);
    int blk_size = atoi(argv[5]);
    int switch_iter = atoi(argv[6]);
	
	//Read graph to CPU
	graph<long, long, long,vertex_t, index_t, weight_t>
		*ginst=new graph<long, long, long,vertex_t, index_t, weight_t>
		(file_beg_pos, file_adj_list, file_weight_list);
	
	feature_t *level, *level_h;
	hipMalloc((void **)&level, sizeof(feature_t));
	hipHostMalloc((void **)&level_h, sizeof(feature_t));
    hipMemset(level, 0, sizeof(feature_t));

	cb_reducer vert_selector_push_h;
	cb_reducer vert_selector_pull_h;
	hipMemcpyFromSymbol(&vert_selector_push_h,HIP_SYMBOL(vert_selector_push_d),sizeof(cb_reducer));
	hipMemcpyFromSymbol(&vert_selector_pull_h,HIP_SYMBOL(vert_selector_pull_d),sizeof(cb_reducer));
	
	cb_mapper vert_behave_push_h;
	cb_mapper vert_behave_pull_h;
	hipMemcpyFromSymbol(&vert_behave_push_h,HIP_SYMBOL(vert_behave_push_d),sizeof(cb_reducer));
	hipMemcpyFromSymbol(&vert_behave_pull_h,HIP_SYMBOL(vert_behave_pull_d),sizeof(cb_reducer));
	
	for(vertex_t st = 0; st < ginst->vert_count; st++) {
		//Init three data structures
		printf("---at node %d---\n", st);
		gpu_graph ggraph(ginst);
		meta_data mdata(ginst->vert_count, ginst->edge_count);
		Barrier global_barrier(BLKS_NUM);
		
		init<<<256,256>>>(st, ginst->vert_count, mdata);
		mapper compute_mapper(ggraph, mdata, vert_behave_push_h, vert_behave_pull_h);
		reducer worklist_gather(ggraph, mdata, vert_selector_push_h, vert_selector_pull_h);
		H_ERR(hipDeviceSynchronize());
		
		double time = wtime();

		//* necessary for high diameter graph, e.g., euro.osm and roadnet.ca
		mapper_merge_push(blk_size, level, ggraph, mdata, compute_mapper, worklist_gather, global_barrier);
		H_ERR(hipDeviceSynchronize());
		
		time = wtime() - time;
		std::cout<<"Total time: "<<time<<" second(s).\n";
		
		hipMemcpy(level_h, level, sizeof(feature_t), hipMemcpyDeviceToHost);	
		std::cout<<"Total iteration: "<<level_h[0]<<"\n";
		
		feature_t *packed_gpu_dist = new feature_t[ginst->vert_count];
		H_ERR(hipMemcpy(packed_gpu_dist, mdata.vert_status, 
				sizeof(feature_t) * ginst->vert_count, hipMemcpyDeviceToHost));

		feature_t *unpacked_gpu_dist = new feature_t[ginst->vert_count];
		vertex_t *unpacked_gpu_route = new vertex_t[ginst->vert_count];
		unpack_cpu_dist(packed_gpu_dist, unpacked_gpu_dist, unpacked_gpu_route, ginst->vert_count);

		feature_t *cpu_dist;
		vertex_t *cpu_routes;
		cpu_sssp<index_t, vertex_t, weight_t, feature_t>
			(cpu_dist, cpu_routes, st, ginst->vert_count, ginst->edge_count, ginst->beg_pos,
			ginst->adj_list, ginst->weight);

		if (memcmp(cpu_dist, unpacked_gpu_dist, sizeof(feature_t) * ginst->vert_count) == 0) {
			printf("Distance result correct\n");
		} else {
			printf("Distance result wrong!\n");
			printf("GPU - CPU\n");
			for(int i = 0; i < ginst->vert_count; i ++) {
				if(unpacked_gpu_dist[i] != cpu_dist[i]) {
					printf("%d: %d - %d\n", i, unpacked_gpu_dist[i], cpu_dist[i]);
				}
			}
			break;
		}
		delete[]  packed_gpu_dist;
		delete[]  unpacked_gpu_dist;
		delete[]  unpacked_gpu_route;
		delete[] cpu_dist;
		mdata.free_md();
	}
}
