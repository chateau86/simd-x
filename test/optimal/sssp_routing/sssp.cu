#include "hip/hip_runtime.h"
#include "header.h"
#include "util.h"
#include "mapper.cuh"
#include "reducer.cuh"
#include "wtime.h"
#include "barrier.cuh"
#include "gpu_graph.cuh"
#include "meta_data.cuh"
#include "mapper_enactor.cuh"
#include "reducer_enactor.cuh"
#include "cpu_sssp_route.hpp"

/*user defined vertex behavior function*/
__inline__ __host__ __device__ data_return_t user_mapper_push(
    vertex_t 	src,
    vertex_t	dest,
    feature_t	level,
    index_t*	beg_pos,
    weight_t	edge_weight,
    feature_t* vert_status,
    feature_t* vert_status_prev
){
	data_return_t dat;
	dat.feature = vert_status[src] + edge_weight;
	dat.data_out = src;
	//dat.data_out = -9999;
	return dat;
} 

/*user defined vertex behavior function*/
__inline__ __host__ __device__ bool vertex_selector_push(
    vertex_t 		vert_id, 
    feature_t 		level,
    vertex_t 		*adj_list, 
    index_t 		*beg_pos, 
    feature_t* 	vert_status,
    feature_t* 	vert_status_prev
){
	return (vert_status[vert_id] != vert_status_prev[vert_id]);
}

/*user defined vertex behavior function*/
__inline__ __host__ __device__ data_return_t user_mapper_pull(
    vertex_t 	src,
    vertex_t	dest,
    feature_t	level,
    index_t*	beg_pos,
    weight_t	edge_weight,
    feature_t* vert_status,
    feature_t* vert_status_prev
){
	// NOT USED HERE
	//return vert_status[src] + edge_weight;
	data_return_t dat;
	dat.feature = vert_status[src] + edge_weight;
	//dat.data_out = dest;
	dat.data_out = -999;
	return dat;
}

/*user defined vertex behavior function*/
__inline__ __host__ __device__ bool vertex_selector_pull(
    vertex_t 	vert_id, 
    feature_t 	level,
    vertex_t* 	adj_list, 
    index_t* 	beg_pos, 
    feature_t* vert_status,
    feature_t* vert_status_prev
){
	return true;
}

__device__ cb_reducer vert_selector_push_d = vertex_selector_push;
__device__ cb_reducer vert_selector_pull_d = vertex_selector_pull;
__device__ cb_mapper vert_behave_push_d = user_mapper_push;
__device__ cb_mapper vert_behave_pull_d = user_mapper_pull;


/*init sssp*/
__global__ void
init(vertex_t src_v, vertex_t vert_count, meta_data mdata)
{
	index_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	while(tid < vert_count) {
		if(tid != src_v) {
			mdata.vert_status[tid] = INFTY;
			mdata.vert_status_prev[tid] = INFTY;
			mdata.vert_data_out[tid] = -1;

		} else {
			mdata.vert_status[tid] = 0;
			mdata.vert_status_prev[tid] = INFTY;
			mdata.vert_data_out[tid] = -1;

			mdata.worklist_mid[0] = src_v;
			mdata.worklist_sz_sml[0] = 0;
			mdata.worklist_sz_mid[0] = 1;
			mdata.worklist_sz_lrg[0] = 0;
			//mdata.bitmap[src_v>>3] |= (1<<(src_v & 7));
		}
		tid += blockDim.x * gridDim.x;
	}
}

int main(int args, char **argv)
{
    // Based on the high-diameter SSSP
	std::cout<<"Input: /path/to/exe /path/to/beg_pos /path/to/adj_list /path/weight_list src blk_size switch_iter\n";
	if(args<5){
        std::cout<<"Wrong input\n";exit(-1);
    }
    for(int i = 0; i < args; i++) {
		std::cout<<argv[i]<<" ";
	}
    std::cout<<"\n";
		
	double tm_map,tm_red,tm_scan;
	char *file_beg_pos = argv[1];
	char *file_adj_list = argv[2];
	char *file_weight_list = argv[3];
	vertex_t src_v = (vertex_t)atol(argv[4]);
    int blk_size = atoi(argv[5]);
    int switch_iter = atoi(argv[6]);
	
	//Read graph to CPU
	graph<long, long, long,vertex_t, index_t, weight_t>
		*ginst=new graph<long, long, long,vertex_t, index_t, weight_t>
		(file_beg_pos, file_adj_list, file_weight_list);
	
	feature_t *level, *level_h;
	hipMalloc((void **)&level, sizeof(feature_t));
	hipHostMalloc((void **)&level_h, sizeof(feature_t));
    hipMemset(level, 0, sizeof(feature_t));

	cb_reducer vert_selector_push_h;
	cb_reducer vert_selector_pull_h;
	hipMemcpyFromSymbol(&vert_selector_push_h,HIP_SYMBOL(vert_selector_push_d),sizeof(cb_reducer));
	hipMemcpyFromSymbol(&vert_selector_pull_h,HIP_SYMBOL(vert_selector_pull_d),sizeof(cb_reducer));
	
	cb_mapper vert_behave_push_h;
	cb_mapper vert_behave_pull_h;
	hipMemcpyFromSymbol(&vert_behave_push_h,HIP_SYMBOL(vert_behave_push_d),sizeof(cb_reducer));
	hipMemcpyFromSymbol(&vert_behave_pull_h,HIP_SYMBOL(vert_behave_pull_d),sizeof(cb_reducer));
	
	//Init three data structures
	gpu_graph ggraph(ginst);
	//meta_data mdata(ginst->vert_count, ginst->edge_count);
    Barrier global_barrier(BLKS_NUM);
    
	//init<<<256,256>>>(src_v, ginst->vert_count, mdata); // Init() does not do any malloc inside (zeroing out only)
	//mapper compute_mapper(ggraph, mdata, vert_behave_push_h, vert_behave_pull_h);
	//reducer worklist_gather(ggraph, mdata, vert_selector_push_h, vert_selector_pull_h);
	H_ERR(hipDeviceSynchronize());
	double total_time = 0;

	//TODO: Multithread this loop


	const size_t FEAT_SZ=sizeof(feature_t)*(ginst->vert_count);
	const size_t DATA_OUT_SZ=sizeof(data_out_cell_t)*(ginst->vert_count)*(ginst->vert_count);
	feature_t *vert_status; // per thread
	feature_t *vert_status_prev; // per thread
	data_out_cell_t *vert_data_out; // global
	H_ERR(hipMalloc((void **)&vert_status, FEAT_SZ));
	H_ERR(hipMalloc((void **)&vert_status_prev, FEAT_SZ));
	H_ERR(hipMalloc((void **)&vert_data_out, DATA_OUT_SZ));
	// vert_data_out is [to_final_node][from_at]
	for(vertex_t src_v = 0; src_v < ginst->vert_count; src_v++) {
		printf("\t\t--- At start node %d ---\n", src_v);
		double time = wtime();

		//* necessary for high diameter graph, e.g., euro.osm and roadnet.ca
		// mapper_merge_push in mapper_enactor.cuh:~930

		//TODO: Multi-stream with pthread
		// https://developer.nvidia.com/blog/gpu-pro-tip-cuda-7-streams-simplify-concurrency/
		// see "A Multi-threading Example"
		// nvcc --default-stream per-thread give one stream per thread with no CUDA work required

		// Kernel _can_ share same block of dev memory https://leimao.github.io/blog/CUDA-Stream/ 
		meta_data mdata(ginst->vert_count, ginst->edge_count, vert_status, vert_status_prev, &(vert_data_out[src_v]));
		// TODO: Metadata pointing to the right instance of vert_status and friends for multithread
		mapper compute_mapper(ggraph, mdata, vert_behave_push_h, vert_behave_pull_h);
		reducer worklist_gather(ggraph, mdata, vert_selector_push_h, vert_selector_pull_h);
		H_ERR(hipDeviceSynchronize());


		init<<<256,256>>>(src_v, ginst->vert_count, mdata); // Init() does not do any malloc inside (zeroing out only)
		mapper_merge_push(blk_size, level, ggraph, mdata, compute_mapper, worklist_gather, global_barrier);
		
		//mapper_merge_push() only ever use 1 stream
		
		time = wtime() - time;
		total_time += time;
		std::cout<<"Total time: "<<time<<" second(s).\n";
		
		hipMemcpy(level_h, level, sizeof(feature_t), hipMemcpyDeviceToHost);	
		std::cout<<"Total iteration: "<<level_h[0]<<"\n";
		
		feature_t *gpu_dist = new feature_t[ginst->vert_count];
		feature_t *gpu_routes = new data_out_cell_t[ginst->vert_count];
		hipMemcpy(gpu_dist, mdata.vert_status, 
				sizeof(feature_t) * ginst->vert_count, hipMemcpyDeviceToHost);
		hipMemcpy(gpu_routes, mdata.vert_data_out, 
				sizeof(data_out_cell_t) * ginst->vert_count, hipMemcpyDeviceToHost);
		//if(src_v % 50 == 0) {
		if(true) {
			feature_t *cpu_dist;
			data_out_cell_t *cpu_routes;
			cpu_sssp<index_t, vertex_t, weight_t, feature_t>
				(cpu_dist, cpu_routes, src_v, ginst->vert_count, ginst->edge_count, ginst->beg_pos,
				ginst->adj_list, ginst->weight);
			if (memcmp(cpu_dist, gpu_dist, sizeof(feature_t) * ginst->vert_count) == 0) {
				printf("Distance result correct\n");
				//Now check route
				if (memcmp(cpu_routes, gpu_routes, sizeof(data_out_cell_t) * ginst->vert_count) == 0) {
					printf("Route result correct\n");
				}else{
					printf("Route result wrong!\n");
					//TODO: "deep inspect" route by traversing back to root and check weight
					printf("GPU - CPU\n");
					for(int i = 0; i < 10; i ++) {
						std::cout<<gpu_routes[i]<<" "<<cpu_routes[i]<<"\n";
					}
					break;
				}

			} else {
				printf("Distance result wrong!\n");
				for(int i = 0; i < 10; i ++) {
					std::cout<<gpu_dist[i]<<" "<<cpu_dist[i]<<"\n";
				}
				break;
			}
			delete[] cpu_dist;
		}
		delete[] gpu_dist;

		mdata.free_md();
	}
	std::cout<<"Total APSP time: "<<total_time<<" second(s).\n";

}
